#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define TILE_WIDTH 16
__constant__ float const_mem_matrix[15000];

__global__ void conv_forward_kernel(float *output, const float *input, const float *mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    Batch - batch_size (number of images in x)
    Map_out - number of output feature maps
    Channel - number of input feature maps
    Height - input height dimension
    Width - input width dimension
    K - kernel height and width (K x K)
    */

    const int Height_out = Height - K + 1;// Define output height and width
    const int Width_out = Width - K + 1;
    
    const int Width_grid = (Width_out + TILE_WIDTH - 1) / TILE_WIDTH; // Compute number of thread blocks needed to cover the output tensor

    // (void)Height_out; // silence declared but never referenced warning. remove this line when you start working
    // (void)Width_out; // silence declared but never referenced warning. remove this line when you start working

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // out_4d(0,0,0,0) = a

    
#define out_4d(i3, i2, i1, i0) output[(i3) * (Map_out * Height_out * Width_out) + (i2) * (Height_out * Width_out) + (i1) * (Width_out) + i0] // Define macros to simplify indexing of tensors
#define in_4d(i3, i2, i1, i0) input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
#define mask_4d(i3, i2, i1, i0) const_mem_matrix[(i3) * (Channel * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    // Insert your GPU convolution kernel code here

    int H = (blockIdx.y / Width_grid) * TILE_WIDTH + threadIdx.y; // Compute the index of the pixel to be computed by the current thread
    int W = (blockIdx.y % Width_grid) * TILE_WIDTH + threadIdx.x;

    // Check if the thread is within the bounds of the output tensor
    if (H < Height_out && W < Width_out){ // for all height and width pixel values
        float inter = 0.0f; // declaring a temp variable
        for (int C = 0; C < Channel; C++) { // sum over all channels
            for (int k = 0; k < K; k++){ // loop over KxK filter
                for (int i = 0; i < K; i++){
                    // Compute the dot product of the input tensor and filter kernel
                    inter += in_4d(blockIdx.z, C, H + k, W + i) * mask_4d(blockIdx.x, C, k, i); // calculating convolution and adding the intermediate results to inter variable
                }
            }
        }
        out_4d(blockIdx.z, blockIdx.x, H, W) = inter; // storing the final results in out_4d
    }

    #undef out_4d // Undefine macros
    #undef in_4d
    #undef mask_4d
}

	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;

    int out_size = ((Height_out*Width_out) * Map_out * Batch) * sizeof(float); // output size is batchsize * output channels * size of each output image
    int in_size = (Height*Width) * Channel * Batch * sizeof(float); // input size is input image dimensions * channels * batchsize
    int k_size = (K*K) * Map_out * Channel * sizeof(float); //each filter times input channels and output feature maps

    hipMalloc((void**)device_input_ptr, in_size);
    // hipMalloc((void**)device_mask_ptr, k_size);
    hipMalloc((void**)device_output_ptr, out_size);
    

    hipMemcpy(*device_input_ptr, host_input, in_size, hipMemcpyHostToDevice);
    // hipMemcpy(*device_mask_ptr, host_mask, k_size, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(const_mem_matrix), host_mask, k_size);

}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Set the kernel dimensions and call the kernel
    const int Height_ = ((Height - K + 1) + TILE_WIDTH - 1) / TILE_WIDTH; // Calculate the dimensions of the kernel block and grid
    const int Width_ = ((Width - K + 1) + TILE_WIDTH - 1) / TILE_WIDTH;

    dim3 blockDim(TILE_WIDTH, TILE_WIDTH, 1); // Set the dimensions of the kernel block and grid
    dim3 gridDim(Map_out, Width_ * Height_ , Batch);

    conv_forward_kernel<<<gridDim, blockDim>>>(device_output, device_input, device_mask, Batch, Map_out, Channel, Height, Width, K);

}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;

    int out_size = (Height_out*Width_out) * Map_out * Batch * sizeof(float);

    // Copy the output back to host

    hipMemcpy(host_output, device_output, out_size, hipMemcpyDeviceToHost);

    // Free device memory

    hipFree(device_output);
    hipFree(device_input);
    // hipFree(device_mask);
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
